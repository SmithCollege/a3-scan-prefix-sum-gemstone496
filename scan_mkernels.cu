
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 512
#define BLOCK_SIZE 64
#define RANGE 4

// function to calculate the scan on GPU
__global__ void scan_range(int *in, int *out, int *sums){
  int gindex = threadIdx.x + blockIdx.x*blockDim.x;
  int start = gindex*RANGE;
  int end = start+RANGE < SIZE ? start+RANGE : SIZE;
  
  int sum = 0;
  for (int i = start; i < end; i++) {
    sum += in[i];
    out[i] = sum;
  }

  int sumsLen = (SIZE+RANGE-1)/RANGE; // add in appropriate sums
  for (int i = gindex+1; i < sumsLen; i++){
    sums[gindex] += sum;
  }
}

__global__ void scan_final(int *out, int *sums){
  int gindex = threadIdx.x + blockIdx.x*blockDim.x;
  int sindex = gindex/RANGE;

  out[gindex] += sums[sindex];
}

int main() {
  // allocate input and output arrays
  int *in; hipMallocManaged(&in, SIZE*sizeof(int)); //these belong on the same lines bc they're var assignment:
  int *out; hipMallocManaged(&out, SIZE*sizeof(int)); //on a technicality, the statements must be separated.
  int *sums; hipMallocManaged(&sums, (SIZE+RANGE-1)/RANGE *sizeof(int)); //ceiling of SIZE/RANGE
  
  // initialize inputs
  for (int i = 0; i < SIZE; i++) {
    in[i] = 1;
  }
  for (int i = 0; i < (SIZE+RANGE-1)/RANGE; i++) {
    sums[i] = 0;
  }

  // do the scan
  int numBlocks = (SIZE + BLOCK_SIZE*RANGE - 1) / (BLOCK_SIZE*RANGE);
  scan_range<<< numBlocks, BLOCK_SIZE >>>(in, out, sums);
  hipDeviceSynchronize(); // patience, girls
  scan_final<<< numBlocks, BLOCK_SIZE*RANGE >>>(out, sums);
  hipDeviceSynchronize(); // remain patient

  // check results
  for (int i = 0; i < SIZE; i++) {
    int ans = i+1;
    out[i] == ans ? printf("%d ", out[i]) : printf("\n  IDX: %d   OUT: %d   EXP: %d\n", i, out[i], ans);
  }
  printf("\n");

  // free mem
  hipFree(in);
  hipFree(out);

  return 0;
}
