
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>

#define SIZE 2048
#define BLOCK_SIZE 128
#define RANGE 16
#define RUNS 100

// function to calculate the scan on GPU
__global__ void scan_range(int *in, int *out, int *sums){
  int gindex = threadIdx.x + blockIdx.x*blockDim.x;
  int start = gindex*RANGE;
  int end = start+RANGE < SIZE ? start+RANGE : SIZE;
  
  int sum = 0;
  for (int i = start; i < end; i++) {
    sum += in[i];
    out[i] = sum;
  }

  int sumsLen = (SIZE+RANGE-1)/RANGE; // length of sums, so we don't edit oob
  ++gindex < sumsLen ? sums[gindex] += sum : sums[gindex] = 0; // it's ok it'll work i pwomise
    //printf("idx: %d   sum: %d   \n", gindex, sum);
    //for (++gindex; gindex < sumsLen; gindex++){
    //sums[gindex] += sum;
    //}
}

__global__ void scan_final(int *out, int *sums){
  int gindex = threadIdx.x + blockIdx.x*blockDim.x;
  int sindex = gindex/RANGE;

  out[gindex] += sums[sindex];
}

int main() {
  // allocate input and output arrays
  int sumsLen = (SIZE+RANGE-1)/RANGE; //ceiling of SIZE/RANGE
  int numBlocks = (SIZE + BLOCK_SIZE*RANGE - 1) / (BLOCK_SIZE*RANGE);
  int *in; hipMallocManaged(&in, SIZE*sizeof(int)); //these belong on the same lines bc they're var assignment:
  int *out; hipMallocManaged(&out, SIZE*sizeof(int)); //on a technicality, the statements must be separated.
  int *sums; hipMallocManaged(&sums, sumsLen*sizeof(int));
  
  
  // initialize inputs
  for (int i = 0; i < SIZE; i++) {
    in[i] = 1;
  }
  for (int i = 0; i < sumsLen; i++) {
    sums[i] = 0;
  }

  for (int i = 0; i < RUNS; i++) {
    int cSum = 0;
    const auto start{std::chrono::steady_clock::now()};
    
    scan_range<<< numBlocks, BLOCK_SIZE >>>(in, out, sums);
    hipDeviceSynchronize(); // patience, girls
    for (int i = 1; i < sumsLen; i++) {
      sums[i] += cSum;
      cSum = sums[i];
    }
    scan_final<<< numBlocks, BLOCK_SIZE*RANGE >>>(out, sums);
    hipDeviceSynchronize(); // remain patient
    
    const auto end{std::chrono::steady_clock::now()};
    const std::chrono::duration<double> elapsed{end - start};
    std::cout << elapsed.count() << "\n";
  }
  
  // do the scan
  scan_range<<< numBlocks, BLOCK_SIZE >>>(in, out, sums);
  hipDeviceSynchronize(); // patience, girls

  int cSum = 0;
  for (int i = 1; i < sumsLen; i++) {
    sums[i] += cSum;
    cSum = sums[i];
  }
  
  scan_final<<< numBlocks, BLOCK_SIZE*RANGE >>>(out, sums);
  hipDeviceSynchronize(); // remain patient

  // check results
  for (int i = 0; i < SIZE; i++) {
    int ans = i+1;
    if (out[i] != ans) { std::cerr << "IDX: " << i << "   OUT: " << out[i] << "   EXP: " << ans << std::endl; }
  }

  // free mem
  hipFree(in);
  hipFree(out);

  return 0;
}
