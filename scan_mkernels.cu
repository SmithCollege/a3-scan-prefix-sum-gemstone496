
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>

#define SIZE 16384
#define BLOCK_SIZE 128
#define RANGE 8
#define RUNS 100

// function to calculate the scan on GPU
__global__ void scan_range(int *in, int *out, int *sums){
  int gindex = threadIdx.x + blockIdx.x*blockDim.x;
  int start = gindex*RANGE;
  int end = start+RANGE < SIZE ? start+RANGE : SIZE;
  
  int sum = 0;
  for (int i = start; i < end; i++) {
    sum += in[i];
    out[i] = sum;
  }

  int sumsLen = (SIZE+RANGE-1)/RANGE; // length of sums, so we don't edit oob
  if (gindex+1 < sumsLen) { sums[gindex+1] += sum; } // it's ok it'll work i pwomise
    //for (++gindex; gindex < sumsLen; gindex++){
    //sums[gindex] += sum;
    //}
}

__global__ void scan_final(int *out, int *sums){
  int gindex = threadIdx.x + blockIdx.x*blockDim.x;
  int sindex = gindex/RANGE;

  out[gindex] += sums[sindex];
}

int main() {
  std::cout << "\n" << SIZE;
  
  // allocate input and output arrays
  int sumsLen = (SIZE+RANGE-1)/RANGE; //ceiling of SIZE/RANGE
  int numBlocks = (sumsLen+BLOCK_SIZE-1) / BLOCK_SIZE;
  int *in; hipMallocManaged(&in, SIZE*sizeof(int)); //these belong on the same lines bc they're var assignment:
  int *out; hipMallocManaged(&out, SIZE*sizeof(int)); //on a technicality, the statements must be separated.
  int *sums; hipMallocManaged(&sums, sumsLen*sizeof(int));
  
  
  // initialize inputs
  for (int i = 0; i < SIZE; i++) {
    in[i] = 1;
  }
  for (int i = 0; i < sumsLen; i++) {
    sums[i] = 0;
  }

  for (int i = 0; i < RUNS; i++) {
    // initialize inputs
    for (int j = 0; j < SIZE; j++) {
      in[j] = 1;
    }
    for (int j = 0; j < sumsLen; j++) {
      sums[j] = 0;
    }
    
    const auto start{std::chrono::steady_clock::now()};

    int cSum = 0;
    scan_range<<< numBlocks, BLOCK_SIZE >>>(in, out, sums);
    hipDeviceSynchronize(); // patience, girls
    for (int i = 1; i < sumsLen; i++) {
      sums[i] += cSum;
      cSum = sums[i];
    }
    scan_final<<< numBlocks, BLOCK_SIZE*RANGE >>>(out, sums);
    hipDeviceSynchronize(); // remain patient
    
    const auto end{std::chrono::steady_clock::now()};
    const std::chrono::duration<double> elapsed{end - start};
    std::cout << "," << elapsed.count();
  }
  
  // check results
  for (int i = 0; i < SIZE; i++) {
    int ans = i+1;
    if (out[i] != ans) { std::cerr << "IDX: " << i << "   OUT: " << out[i] << "   EXP: " << ans << std::endl; }
  }

  // free mem
  hipFree(in);
  hipFree(out);

  return 0;
}
