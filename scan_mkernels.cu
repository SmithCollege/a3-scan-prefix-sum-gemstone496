
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 512
#define BLOCK_SIZE 64
#define RANGE 4

// function to calculate the scan on GPU
__global__ void scan_range(int *in, int *out, int *sums){
  int gindex = threadIdx.x + blockIdx.x*blockDim.x;
  int start = gindex*RANGE;
  int end = start+RANGE < SIZE ? start+RANGE : SIZE;
  
  int sum = 0;
  for (int i = start; i < end; i++) {
    sum += in[i];
    out[i] = sum;
  }

  int sumsLen = (SIZE+RANGE-1)/RANGE; // length of sums, so we don't edit oob
  ++gindex < sumsLen ? sums[gindex] += sum : sums[gindex] = 0; // it's ok it'll work i pwomise
    //printf("idx: %d   sum: %d   \n", gindex, sum);
    //for (++gindex; gindex < sumsLen; gindex++){
    //sums[gindex] += sum;
    //}
}

__global__ void scan_final(int *out, int *sums){
  int gindex = threadIdx.x + blockIdx.x*blockDim.x;
  int sindex = gindex/RANGE;

  out[gindex] += sums[sindex];
}

int main() {
  // allocate input and output arrays
  int sumsLen = (SIZE+RANGE-1)/RANGE; //ceiling of SIZE/RANGE
  int *in; hipMallocManaged(&in, SIZE*sizeof(int)); //these belong on the same lines bc they're var assignment:
  int *out; hipMallocManaged(&out, SIZE*sizeof(int)); //on a technicality, the statements must be separated.
  int *sums; hipMallocManaged(&sums, sumsLen*sizeof(int)); 
  
  
  // initialize inputs
  for (int i = 0; i < SIZE; i++) {
    in[i] = 1;
  }
  for (int i = 0; i < sumsLen; i++) {
    sums[i] = 0;
  }

  // do the scan
  int numBlocks = (SIZE + BLOCK_SIZE*RANGE - 1) / (BLOCK_SIZE*RANGE);
  scan_range<<< numBlocks, BLOCK_SIZE >>>(in, out, sums);
  hipDeviceSynchronize(); // patience, girls

  int cSum = 0;
  for (int i = 1; i < sumsLen; i++) {
    sums[i] += cSum;
    cSum = sums[i];
  }
  
  scan_final<<< numBlocks, BLOCK_SIZE*RANGE >>>(out, sums);
  hipDeviceSynchronize(); // remain patient

  // check results
  for (int i = 0; i < SIZE; i++) {
    int ans = i+1;
    out[i] == ans ? printf("%d ", out[i]) : printf("\n  IDX: %d   OUT: %d   EXP: %d\n", i, out[i], ans);
  }
  printf("\n");

  // free mem
  hipFree(in);
  hipFree(out);

  return 0;
}
