#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define SIZE 512
#define BLOCK_SIZE 64
#define RANGE 4

// function to calculate the scan on GPU
__global__ void scan_range(int *in, int *out, int*sums){
  int gindex = threadIdx.x + blockIdx.x*blockDim.x;
  int start = gindex*RANGE;
  int end = start+RANGE < SIZE ? start+RANGE : SIZE;
  
  int sum = 0;
  for (int i = start; i < end; i++) {
    sum += in[i];
    out[i] = sum;
  }

  int sumsLen = (SIZE+RANGE-1)/RANGE; // add in 
  for (int i = gindex; i < sumsLen; i++){
    sums[gindex] += sum;
  }
}

int main() {
  // allocate input and output arrays
  int *in; hipMallocManaged(&in, SIZE*sizeof(int)); //these belong on the same lines bc they're var assignment:
  int *out; hipMallocManaged(&out, SIZE*sizeof(int)); //on a technicality, the statements must be separated.
  int *sums; cudaMallocmanaged(&sums, (SIZE+RANGE-1)/RANGE *sizeof(int)); //ceiling of SIZE/RANGE
  
  // initialize inputs
  for (int i = 0; i < SIZE; i++) {
    in[i] = 1;
  }
  for (int i = 0; i < (SIZE+RANGE-1)/RANGE; i++) {
    sums[i] = 0;
  }

  // do the scan
  int numBlocks = (SIZE + BLOCK_SIZE*RANGE - 1) / (BLOCK_SIZE*RANGE);
  scan_range<<< numBlocks, BLOCK_SIZE >>>(in, out, sums);
  hipDeviceSynchronize(); // patience, girls
  scan_finish<<< numBlocks, BLOCK_SIZE*RANGE >>>(in, out, sums);
  hipDeviceSynchronize();

  // check results
  for (int i = 0; i < SIZE; i++) {
    int ans = i+1;
    out[i] == ans ? printf("%d ", out[i]) : printf("\n  IDX: %d   OUT: %d   EXP: %d\n", i, out[i], ans);
  }
  printf("\n");

  // free mem
  hipFree(in);
  hipFree(out);

  return 0;
}
