
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>

#define SIZE 16384
#define BLOCK_SIZE 256
#define RUNS 100

// function to calculate the scan on GPU
__global__ void scan(int *in, int *out){  
  int gindex = threadIdx.x + blockIdx.x*blockDim.x;
  int end = gindex < SIZE ? gindex+1 : SIZE;
  
  int sum = 0;
  for (int i = 0; i < end; i++) {
    sum += in[i];
  }

  out[gindex] = sum;
}

int main() {
  std::cout << "\n" << SIZE;
  
  // allocate input and output arrays
  int *in; hipMallocManaged(&in, SIZE*sizeof(int)); //these belong on the same lines bc they're var assignment:
  int *out; hipMallocManaged(&out, SIZE*sizeof(int)); //on a technicality, the statements must be separated.
  
  for (int i = 0; i < RUNS; i++) {
    // initialize inputs
    for (int j = 0; j < SIZE; j++) {
      in[j] = 1;
    }
    
    const auto start{std::chrono::steady_clock::now()};
    scan<<< (SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE , BLOCK_SIZE >>>(in, out);
    hipDeviceSynchronize(); // patience, girls
    const auto end{std::chrono::steady_clock::now()};
    const std::chrono::duration<double> elapsed{end - start};
    std::cout << "," << elapsed.count();
  }

  // check results
  for (int i = 0; i < SIZE; i++) {
    int ans = i+1;
    if (out[i] != ans) { std::cerr << "IDX: " << i << "   OUT: " << out[i] << "   EXP: " << ans << std::endl; }
  }

  // free mem
  hipFree(in);
  hipFree(out);

  return 0;
}
