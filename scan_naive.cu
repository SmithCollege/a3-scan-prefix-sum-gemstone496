
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 512
#define BLOCK_SIZE 64

// function to calculate the scan on GPU
__global__ void scan(int *in, int *out){
  int gindex = threadIdx.x + blockIdx.x*blockDim.x;
  int end = gindex < SIZE ? gindex+1 : SIZE;
  
  int sum = 0;
  for (int i = 0; i < end; i++) {
    sum += in[i];
  }

  out[gindex] = sum;
}

int main() {
  // allocate input and output arrays
  int *in; hipMallocManaged(&in, SIZE*sizeof(int)); //these belong on the same lines bc they're var assignment:
  int *out; hipMallocManaged(&out, SIZE*sizeof(int)); //on a technicality, the statements must be separated.
  
  // initialize inputs
  for (int i = 0; i < SIZE; i++) {
    in[i] = 1;
  }

  // do the scan
  scan<<< (SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE , BLOCK_SIZE >>>(in, out);
  hipDeviceSynchronize(); // patience, girls

  // check results
  for (int i = 0; i < SIZE; i++) {
    int ans = i+1;
    out[i] == ans ? printf("%d ", out[i]) : printf("\n  IDX: %d   OUT: %d   EXP: %d\n", i, out[i], ans);
  }
  printf("\n");

  // free mem
  hipFree(in);
  hipFree(out);

  return 0;
}
