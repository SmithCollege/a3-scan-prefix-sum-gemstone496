
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>

#define SIZE 2048
#define BLOCK_SIZE 128
#define RUNS 100

// function to calculate the scan on GPU
__global__ void scan(int *in, int *out){
  int gindex = threadIdx.x + blockIdx.x*blockDim.x;
  int end = gindex < SIZE ? gindex+1 : SIZE;
  
  int sum = 0;
  for (int i = 0; i < end; i++) {
    sum += in[i];
  }

  out[gindex] = sum;
}

int main() {
  // allocate input and output arrays
  int *in; hipMallocManaged(&in, SIZE*sizeof(int)); //these belong on the same lines bc they're var assignment:
  int *out; hipMallocManaged(&out, SIZE*sizeof(int)); //on a technicality, the statements must be separated.
    
  // initialize inputs
  for (int i = 0; i < SIZE; i++) {
    in[i] = 1;
  }

  for (int i = 0; i < RUNS; i++) {
    const auto start{std::chrono::steady_clock::now()};
    scan<<< (SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE , BLOCK_SIZE >>>(in, out);
    hipDeviceSynchronize(); // patience, girls
    const auto end{std::chrono::steady_clock::now()};
    const std::chrono::duration<double> elapsed{end - start};
    std::cout << elapsed.count() << "\n";
  }

  // check results
  for (int i = 0; i < SIZE; i++) {
    int ans = i+1;
    if (out[i] != ans) { std::cerr << "IDX: " << i << "   OUT: " << out[i] << "   EXP: " << ans << std::endl; }
  }

  // free mem
  hipFree(in);
  hipFree(out);

  return 0;
}
